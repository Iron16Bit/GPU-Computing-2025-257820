
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__
void spmv(int *Arows, int *Acols, double *Avals, double *v, double *C, int rows, int cols, int values) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < values) {
        double product = Avals[tid] * v[Acols[tid]];
        atomicAdd(&C[Arows[tid]], product);
    }
}

// Compute bandwidth and flops
void compute_band_gflops(int rows, int cols, int values, double time) {
    size_t bytes = sizeof(double) * (values + rows + cols) + sizeof(int) * (2 * values);
    double bandwidth = (bytes * 1e-9) / (time * 1e-3);
    double flops = (2 * values) / (time * 1e-3) * 1e-9;
    printf("Bandwidth: %f GB/s\n", bandwidth);
    printf("FLOPS: %f GFLOPS\n", flops);
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f ", a[i]);
    }
    printf("\n");
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%f ", m[i*cols+j]);
        }
        printf("\n");
    }
}

#define ITERATIONS 11
#define DEFAULT_THREADS_PER_BLOCK 256

int main(int argc, char *argv[]) {
    if (argc < 2 || argc > 3) {
        fprintf(stderr, "Usage: %s <input_file> [threads_per_block]\n", argv[0]);
        return 1;
    }

    // Parse threads per block parameter
    int threadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
    if (argc == 3) {
        int user_threads = atoi(argv[2]);
        if (user_threads > 0) {
            threadsPerBlock = user_threads;
        } else {
            fprintf(stderr, "Warning: Invalid threads per block value, using default (%d)\n", 
                    DEFAULT_THREADS_PER_BLOCK);
        }
    }
    
    printf("Using %d threads per block\n", threadsPerBlock);
    printf("Used matrix: %s\n", argv[1]);

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;
    double totalTime = 0.0;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use cudaMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Create dense vector using cudaMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using cudaMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));
    
    // Perform SpMV
    int N = values;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;

    first = 1;

    for (int i=0; i<ITERATIONS; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);

        spmv<<<blocksPerGrid, threadsPerBlock>>>(Arows, Acols, Avals, v, C, rows, cols, values);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        // printf("Kernel completed in %fms\n", e_time);
        if (first == 1) {
            first = 0;
        } else {
            totalTime += e_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    // Calculate average time
    double avg_time = totalTime / (ITERATIONS - 1);
    printf("Average time: %fms\n", avg_time);
    compute_band_gflops(rows, cols, values, avg_time);

    fclose(fin);
    
    // Free using cudaFree instead of free
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);
    hipFree(v);
    hipFree(C);

    return 0;
}
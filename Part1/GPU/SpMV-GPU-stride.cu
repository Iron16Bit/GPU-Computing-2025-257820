
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__
void spmv(int *Arows, int *Acols, double *Avals, double *v, double *C, int rows, int cols, int values) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < values) {
        int total_threads = gridDim.x * blockDim.x; // We stride of gridSize * blockSize

        for (int i=tid; i<values; i+=total_threads) {
            double product = Avals[i] * v[Acols[i]];
            atomicAdd(&C[Arows[i]], product);
        }
    }
}

// Compute bandwidth and flops
void compute_band_gflops(int rows, int cols, int values, double time_ms, int BLOCKS, int THREADS) {
    // 2 floating-point operations per non-zero element (multiply + add)
    double operations = 2.0 * values;
    
    // Convert to GFLOPS: operations / (time in seconds) / 1e9
    double gflops = operations / (time_ms / 1000.0) / 1e9;
    
    // Bandwidth calculation
    size_t bytes = sizeof(double) * (values + rows + cols) + sizeof(int) * (2 * values);
    int total_threads = BLOCKS * THREADS;
    double bandwidth = ((bytes * total_threads) / 1e9) / (time_ms / 1000.0);
    
    printf("Bandwidth: %f GB/s\n", bandwidth);
    printf("FLOPS: %f GFLOPS\n", gflops);
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f ", a[i]);
    }
    printf("\n");
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%f ", m[i*cols+j]);
        }
        printf("\n");
    }
}

#define ITERATIONS 51
#define DEFAULT_THREADS 256
#define DEFAULT_BLOCKS 4

int main(int argc, char *argv[]) {
    if (argc < 2 || argc > 4) {
        fprintf(stderr, "Usage: %s <input_file> [num_threads] [num_blocks]\n", argv[0]);
        return 1;
    }

    int THREADS = DEFAULT_THREADS;
    int BLOCKS = DEFAULT_BLOCKS;

    // Parse threads parameter
    if (argc >= 3) {
        int user_threads = atoi(argv[2]);
        if (user_threads > 0) {
            THREADS = user_threads;
        } else {
            fprintf(stderr, "Warning: Invalid number of threads, using default (%d)\n", DEFAULT_THREADS);
        }
    }

    // Parse blocks parameter
    if (argc >= 4) {
        int user_blocks = atoi(argv[3]);
        if (user_blocks > 0) {
            BLOCKS = user_blocks;
        } else {
            fprintf(stderr, "Warning: Invalid number of blocks, using default (%d)\n", DEFAULT_BLOCKS);
        }
    }

    printf("Using configuration: %d threads per block, %d blocks\n", THREADS, BLOCKS);

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;
    double totalTime = 0.0;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use cudaMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Create dense vector using cudaMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using cudaMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));

    hipEvent_t start, stop;

    first = 1;

    for (int i=0; i<ITERATIONS; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);

        spmv<<<BLOCKS, THREADS>>>(Arows, Acols, Avals, v, C, rows, cols, values);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        // printf("Kernel completed in %fms\n", e_time);
        if (first == 1) {
            first = 0;
        } else {
            totalTime += e_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    printf("Used matrix: %s\n", argv[1]);
    // Calculate average time
    double avg_time = totalTime / (ITERATIONS - 1);
    printf("Average time: %fms\n", avg_time);
    compute_band_gflops(rows, cols, values, avg_time, BLOCKS, THREADS);

    fclose(fin);
    
    // Free using cudaFree instead of free
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);
    hipFree(v);
    hipFree(C);

    return 0;
}

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__
void spmv(int *Arows, int *Acols, double *Avals, double *v, double *C, int rows, int cols, int values) {
    int current_row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (current_row < rows) {
        double sum = 0.0;
        
        // Binary search to find the starting position for this row
        int left = 0;
        int right = values - 1;
        int start_pos = values; // Default to end if row not found
        
        while (left <= right) {
            int mid = left + (right - left) / 2;
            if (Arows[mid] < current_row) {
                left = mid + 1;
            } else if (Arows[mid] > current_row) {
                right = mid - 1;
            } else {
                // Found a match, but we need to find the first occurrence
                start_pos = mid;
                right = mid - 1;
            }
        }
        
        // If no exact match found and left is valid, use it as start point
        if (start_pos == values || Arows[start_pos] != current_row) {
            start_pos = left;
        }
        
        // Accumulate products for this row
        for (int i = start_pos; i < values && Arows[i] == current_row; i++) {
            sum += Avals[i] * v[Acols[i]];
        }
        
        C[current_row] = sum;
    }
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f ", a[i]);
    }
    printf("\n");
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%f ", m[i*cols+j]);
        }
        printf("\n");
    }
}

// double calculateBandwidthGBs(int values, int rows, int cols, double timeMs) {
//     double COO_size = values * (sizeof(int) + sizeof(int) + sizeof(double)); // COO size in bytes
//     double vector_size = cols * sizeof(double); // Dense vector size in bytes
//     double output_size = rows * sizeof(double); // Output vector size in bytes
//     double bytesAccessed = COO_size + vector_size + output_size;

//     // Convert ms to seconds and bytes to GB
//     double timeS = timeMs * 1e-3;
//     double dataGB = bytesAccessed * 1e-9;
    
//     return dataGB / timeS;
// }

int ITERATIONS = 51;

int main(int argc, char *argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;
    double totalTime = 0.0;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use cudaMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Create dense vector using cudaMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using cudaMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));
    
    // Perform SpMV
    int N = rows;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    first = 1;

    hipEvent_t start, stop;

    for (int i=0; i<ITERATIONS; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);

        spmv<<<blocksPerGrid, threadsPerBlock>>>(Arows, Acols, Avals, v, C, rows, cols, values);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        printf("Kernel completed in %fms\n", e_time);
        if (first == 1) {
            first = 0;
        } else {
            totalTime += e_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    // Calculate average time
    double avg_time = totalTime / ITERATIONS;
    printf("Average time: %fms\n", avg_time);
    // printf("Bandwidth: %f GB/s\n", calculateBandwidthGBs(values, rows, cols, avg_time));

    fclose(fin);
    
    // Free using cudaFree instead of free
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);
    hipFree(v);
    hipFree(C);

    return 0;
}
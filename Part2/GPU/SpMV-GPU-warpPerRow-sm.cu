
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__
void spmv(int *row_pointer, int *Acols, double *Avals, double *v, double *C, int rows, int cols, int values) {
    int row = blockIdx.x;
    int lane = threadIdx.x;
    
    // Shared memory for row data
    extern __shared__ char shared_mem[];
    int *shared_cols = (int*)shared_mem;
    double *shared_vals = (double*)(shared_mem + 32 * sizeof(int));
    
    if (row < rows && threadIdx.x < 32) {
        double sum = 0.0;
        int start_pos = row_pointer[row];
        int end_pos = row_pointer[row+1];
        int row_nnz = end_pos - start_pos;
        
        // Load row data into shared memory in chunks
        for (int chunk_start = 0; chunk_start < row_nnz; chunk_start += 32) {
            int chunk_size = min(32, row_nnz - chunk_start);
            
            // Cooperatively load chunk into shared memory
            if (lane < chunk_size) {
                int global_idx = start_pos + chunk_start + lane;
                shared_cols[lane] = Acols[global_idx];
                shared_vals[lane] = Avals[global_idx];
            }
            __syncwarp();
            
            // Process chunk from shared memory
            if (lane < chunk_size) {
                sum += shared_vals[lane] * __ldg(&v[shared_cols[lane]]);
            }
            __syncwarp();
        }
        
        // Warp-level reduction with loop unrolling
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
        
        if (lane == 0) {
            C[row] = sum;
        }
    }
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f\n", a[i]);
    }
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows*cols; i++) {
        printf("%f\n", m[i]);
    }
}

// Compute bandwidth and flops
void compute_band_gflops(int rows, int cols, int values, double time_ms, int* Acols) {
    // Bytes read from the CSR
    size_t csr_size = (size_t)(sizeof(int) * values + sizeof(int) * (rows+1) + sizeof(double) * values);
    // Bytes read from the dense vector
    int* unique_cols = (int*)calloc(cols, sizeof(int));
    int unique_count = 0;
    for (int i=0; i<values; i++) {
        if (unique_cols[Acols[i]] == 0) {
            unique_cols[Acols[i]] = 1;
            unique_count += 1;
        }
    }
    size_t vector_size = (size_t)(sizeof(double) * unique_count);
    // Total bytes read
    size_t bytes_read = csr_size + vector_size;
    // Bytes written
    size_t bytes_written = (size_t)(sizeof(double) * rows);
    size_t total_bytes = bytes_read + bytes_written;

    // GFLOPS
    double bandwidth = total_bytes / (time_ms * 1.0e6);
    double operations = 2.0 * values;
    double gflops = operations / (time_ms * 1.0e6);

    printf("Bandwidth: %f GB/s\n", bandwidth);
    printf("FLOPS: %f GFLOPS\n", gflops);
}


void convert_coo_to_csr(int *Arows, int *Acols, double *Avals, 
                        int **csr_cols, double **csr_vals, int *row_pointer, 
                        int values, int rows) {
    // Initialize row pointer
    for (int i = 0; i <= rows; i++) {
        row_pointer[i] = 0;
    }
    
    // Count elements per row with bounds checking
    for (int i = 0; i < values; i++) {
        if (Arows[i] >= 0 && Arows[i] < rows) {
            row_pointer[Arows[i] + 1]++;
        }
    }
    
    // Convert counts to cumulative offsets
    for (int i = 1; i <= rows; i++) {
        row_pointer[i] += row_pointer[i-1];
    }
    
    // Allocate CSR arrays
    hipMallocManaged(csr_cols, values * sizeof(int));
    hipMallocManaged(csr_vals, values * sizeof(double));
    
    // Create temporary copy of row_pointer for insertion
    int *temp_row_ptr = (int*)malloc((rows + 1) * sizeof(int));
    for (int i = 0; i <= rows; i++) {
        temp_row_ptr[i] = row_pointer[i];
    }
    
    // Fill CSR arrays with bounds checking
    for (int i = 0; i < values; i++) {
        int row = Arows[i];
        if (row >= 0 && row < rows) {
            int pos = temp_row_ptr[row]++;
            if (pos < values) {
                (*csr_cols)[pos] = Acols[i];
                (*csr_vals)[pos] = Avals[i];
            }
        }
    }
    
    free(temp_row_ptr);
}

int ITERATIONS = 51;

int main(int argc, char *argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;
    double totalTime = 0.0;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use cudaMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Convert COO to CSR
    int* row_pointer;
    int* csr_cols;
    double* csr_vals;
    hipMallocManaged(&row_pointer, (rows + 1) * sizeof(int));
    
    convert_coo_to_csr(Arows, Acols, Avals, &csr_cols, &csr_vals, row_pointer, values, rows);
    
    // Free original COO arrays
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);

    // Create dense vector using cudaMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using cudaMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));
    
    // Perform SpMV
    int threadsPerBlock = 32;  // One warp per row
    int blocksPerGrid = rows;  // One block per row

    hipEvent_t start, stop;

    for (int i=0; i<ITERATIONS; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);

        int shared_mem_size = 32 * (sizeof(int) + sizeof(double)); // For approach 2
        spmv<<<blocksPerGrid, threadsPerBlock, shared_mem_size>>>(row_pointer, csr_cols, csr_vals, v, C, rows, cols, values);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // Ensure all operations are completed
        hipDeviceSynchronize();
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        if (first == 1) {
            first = 0;
        } else {
            totalTime += e_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    // Calculate average time
    double avg_time = totalTime / (ITERATIONS - 1);
    printf("Average time: %fms\n", avg_time);
    compute_band_gflops(rows, cols, values, avg_time, csr_cols);

    fclose(fin);
    
    // Free using cudaFree instead of free
    hipFree(csr_cols);
    hipFree(csr_vals);
    hipFree(row_pointer);
    hipFree(v);
    hipFree(C);

    return 0;
}
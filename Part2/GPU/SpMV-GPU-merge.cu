
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
    
__global__
void spmv(int *Arows, int *Acols, double *Avals, double *v, double *C, 
                      int values, int rows) {
    extern __shared__ double sdata[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int block_start = blockIdx.x * blockDim.x;
    int block_end = min((blockIdx.x + 1) * blockDim.x, values);
    
    // Load block of data into shared memory
    double local_sum = 0.0;
    int current_row = -1;
    
    if (tid < values) {
        current_row = Arows[tid];
        local_sum = Avals[tid] * __ldg(&v[Acols[tid]]);
    }
    
    sdata[threadIdx.x] = local_sum;
    __syncthreads();
    
    // Unrolled merge within block for same rows
    if (blockDim.x >= 1024) {
        if (threadIdx.x >= 512 && tid < values) {
            int other_tid = tid - 512;
            if (other_tid >= block_start && Arows[other_tid] == current_row) {
                sdata[threadIdx.x] += sdata[threadIdx.x - 512];
            }
        }
        __syncthreads();
    }
    
    if (blockDim.x >= 512) {
        if (threadIdx.x >= 256 && tid < values) {
            int other_tid = tid - 256;
            if (other_tid >= block_start && Arows[other_tid] == current_row) {
                sdata[threadIdx.x] += sdata[threadIdx.x - 256];
            }
        }
        __syncthreads();
    }
    
    if (blockDim.x >= 256) {
        if (threadIdx.x >= 128 && tid < values) {
            int other_tid = tid - 128;
            if (other_tid >= block_start && Arows[other_tid] == current_row) {
                sdata[threadIdx.x] += sdata[threadIdx.x - 128];
            }
        }
        __syncthreads();
    }
    
    if (blockDim.x >= 128) {
        if (threadIdx.x >= 64 && tid < values) {
            int other_tid = tid - 64;
            if (other_tid >= block_start && Arows[other_tid] == current_row) {
                sdata[threadIdx.x] += sdata[threadIdx.x - 64];
            }
        }
        __syncthreads();
    }
    
    if (blockDim.x >= 64) {
        if (threadIdx.x >= 32 && tid < values) {
            int other_tid = tid - 32;
            if (other_tid >= block_start && Arows[other_tid] == current_row) {
                sdata[threadIdx.x] += sdata[threadIdx.x - 32];
            }
        }
        __syncthreads();
    }
    
    // Final warp-level operations (no sync needed within warp)
    if (threadIdx.x >= 16 && tid < values) {
        int other_tid = tid - 16;
        if (other_tid >= block_start && Arows[other_tid] == current_row) {
            sdata[threadIdx.x] += sdata[threadIdx.x - 16];
        }
    }
    
    if (threadIdx.x >= 8 && tid < values) {
        int other_tid = tid - 8;
        if (other_tid >= block_start && Arows[other_tid] == current_row) {
            sdata[threadIdx.x] += sdata[threadIdx.x - 8];
        }
    }
    
    if (threadIdx.x >= 4 && tid < values) {
        int other_tid = tid - 4;
        if (other_tid >= block_start && Arows[other_tid] == current_row) {
            sdata[threadIdx.x] += sdata[threadIdx.x - 4];
        }
    }
    
    if (threadIdx.x >= 2 && tid < values) {
        int other_tid = tid - 2;
        if (other_tid >= block_start && Arows[other_tid] == current_row) {
            sdata[threadIdx.x] += sdata[threadIdx.x - 2];
        }
    }
    
    if (threadIdx.x >= 1 && tid < values) {
        int other_tid = tid - 1;
        if (other_tid >= block_start && Arows[other_tid] == current_row) {
            sdata[threadIdx.x] += sdata[threadIdx.x - 1];
        }
    }
    
    // Write results for row boundaries
    if (tid < values) {
        bool is_boundary = (tid == values - 1) || 
                          (tid + 1 < values && Arows[tid] != Arows[tid + 1]);
        if (is_boundary) {
            atomicAdd(&C[current_row], sdata[threadIdx.x]);
        }
    }
}

// Compute bandwidth and flops
void compute_band_gflops(int rows, int cols, int values, double time_ms, int* Acols) {
    // Bytes read from the COO
    size_t coo_size = (size_t)(sizeof(int) * (2 * values) + sizeof(double) * values);
    // Bytes read from the dense vector
    int* unique_cols = (int*)calloc(cols, sizeof(int));
    int unique_count = 0;
    for (int i=0; i<values; i++) {
        if (unique_cols[Acols[i]] == 0) {
            unique_cols[Acols[i]] = 1;
            unique_count += 1;
        }
    }
    size_t vector_size = (size_t)(sizeof(double) * unique_count);
    // Total bytes read
    size_t bytes_read = coo_size + vector_size;
    // Bytes written
    size_t bytes_written = (size_t)(sizeof(double) * rows);
    size_t total_bytes = bytes_read + bytes_written;

    // GFLOPS
    double bandwidth = total_bytes / (time_ms * 1.0e6);
    double operations = 2.0 * values;
    double gflops = operations / (time_ms * 1.0e6);

    printf("Bandwidth: %f GB/s\n", bandwidth);
    printf("FLOPS: %f GFLOPS\n", gflops);
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f ", a[i]);
    }
    printf("\n");
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%f ", m[i*cols+j]);
        }
        printf("\n");
    }
}

#define ITERATIONS 51
#define DEFAULT_THREADS_PER_BLOCK 256

int main(int argc, char *argv[]) {
    if (argc < 2 || argc > 3) {
        fprintf(stderr, "Usage: %s <input_file> [threads_per_block]\n", argv[0]);
        return 1;
    }

    // Parse threads per block parameter
    int threadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
    if (argc == 3) {
        int user_threads = atoi(argv[2]);
        if (user_threads > 0) {
            threadsPerBlock = user_threads;
        } else {
            fprintf(stderr, "Warning: Invalid threads per block value, using default (%d)\n", 
                    DEFAULT_THREADS_PER_BLOCK);
        }
    }
    
    printf("Using %d threads per block\n", threadsPerBlock);
    printf("Used matrix: %s\n", argv[1]);

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;
    double totalTime = 0.0;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use cudaMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Create dense vector using cudaMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using cudaMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));
    
    // Perform SpMV
    int N = values;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;

    first = 1;

    for (int i=0; i<ITERATIONS; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);

        // When launching the kernel:
        size_t sharedMemSize = threadsPerBlock * sizeof(double);
        spmv<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
            Arows, Acols, Avals, v, C, values, rows);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        // printf("Kernel completed in %fms\n", e_time);
        if (first == 1) {
            first = 0;
        } else {
            totalTime += e_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    // Calculate average time
    double avg_time = totalTime / (ITERATIONS - 1);
    printf("Average time: %fms\n", avg_time);
    compute_band_gflops(rows, cols, values, avg_time, Acols);

    fclose(fin);
    
    // Free using cudaFree instead of free
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);
    hipFree(v);
    hipFree(C);

    return 0;
}
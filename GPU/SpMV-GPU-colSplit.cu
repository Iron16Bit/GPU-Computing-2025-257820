#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__
void spmv(int *Arows, int *Acols, double *Avals, double *v, double *C, int rows, int cols, int values) {
    int current_col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (current_col < cols) {
        for (int i=0; i<values; i++) {
            if (Acols[i] == current_col) {
                double product = Avals[i] * v[current_col];
                atomicAdd(&C[Arows[i]], product);
            } else if (Acols[i] > current_col) {
                break;
            }
        }
    }
}

void print_int_array(int* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}

void print_double_array(double* a, int n) {
    for (int i=0; i<n; i++) {
        printf("%f ", a[i]);
    }
    printf("\n");
}

void print_matrix(double* m, int rows, int cols) {
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%f ", m[i*cols+j]);
        }
        printf("\n");
    }
}

void swap(int* Arows, int* Acols, double* Avals, int i, int j) {
    int tmp_row = Arows[i];
    int tmp_col = Acols[i];
    double tmp_val = Avals[i];

    Arows[i] = Arows[j];
    Acols[i] = Acols[j];
    Avals[i] = Avals[j];

    Arows[j] = tmp_row;
    Acols[j] = tmp_col;
    Avals[j] = tmp_val;
}

void sort(int* Arows, int* Acols, double* Avals, int n) {
    for (int i=0; i<n-1; i++) {
        for (int j=i+1; j<n; j++) {
            if (Acols[i] > Acols[j]) {
                swap(Arows, Acols, Avals, i, j);
            } else if ((Acols[i] == Acols[j]) && (Arows[i] > Arows[j])) {
                swap(Arows, Acols, Avals, i, j);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    FILE *fin = fopen(argv[1], "r");

    if (!fin) {
        perror("Failed to open file");
        return 1;
    }

    char buffer[100];
    int first = 1;

    int rows;
    int cols;
    int values;

    int *Arows;
    int *Acols;
    double *Avals;

    int counter = 0;
    
    // Create COO from file
    while(fgets(buffer, 100, fin)) {
        if (buffer[0] != '%') {
            char *token = strtok(buffer, " ");
            char split_buffer[3][64];
            for (int i = 0; i < 3; i++) {
                if (!token) break;
                strncpy(split_buffer[i], token, 63);
                split_buffer[i][63] = '\0';
                token = strtok(NULL, " ");
            }
            if (first == 1) {
                first = 0;
                rows = atoi(split_buffer[0]);
                cols = atoi(split_buffer[1]);
                values = atoi(split_buffer[2]);
                
                // Use hipMallocManaged instead of malloc
                hipMallocManaged(&Arows, values*sizeof(int));
                hipMallocManaged(&Acols, values*sizeof(int));
                hipMallocManaged(&Avals, values*sizeof(double));
            } else {
                // Matrix Market files are 1-indexed
                int tmp_row = atoi(split_buffer[0])-1;
                int tmp_col = atoi(split_buffer[1])-1;
                double tmp_val = atof(split_buffer[2]);

                Arows[counter] = tmp_row;
                Acols[counter] = tmp_col;
                Avals[counter] = tmp_val;

                counter+=1;
            }
        }
    }

    // Sort COO
    sort(Arows, Acols, Avals, values);

    // Create dense vector using hipMallocManaged
    double *v;
    hipMallocManaged(&v, cols*sizeof(double));
    for (int i=0; i<cols; i++) {
        v[i] = 1.0;
    }

    // Create output vector using hipMallocManaged
    double *C;
    hipMallocManaged(&C, rows*sizeof(double));
    
    // Perform SpMV
    int N = cols;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;

    for (int i=0; i<10; i++) {
        hipMemset(C, 0, rows * sizeof(double));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // Prefetch data to GPU
        int device = -1;
        hipGetDevice(&device);
        hipMemPrefetchAsync(Arows, values*sizeof(int), device, NULL);
        hipMemPrefetchAsync(Acols, values*sizeof(int), device, NULL);
        hipMemPrefetchAsync(Avals, values*sizeof(double), device, NULL);
        hipMemPrefetchAsync(v, cols*sizeof(double), device, NULL);
        hipMemPrefetchAsync(C, rows*sizeof(double), device, NULL);
        
        hipEventRecord(start);

        spmv<<<blocksPerGrid, threadsPerBlock>>>(Arows, Acols, Avals, v, C, rows, cols, values);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // Ensure all operations are completed
        hipDeviceSynchronize();
        
        float e_time = 0;
        hipEventElapsedTime(&e_time, start, stop);
        // print_double_array(C, rows);
        printf("Kernel completed in %fms\n", e_time);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    // print_double_array(C, rows);

    fclose(fin);
    
    // Free using hipFree instead of free
    hipFree(Arows);
    hipFree(Acols);
    hipFree(Avals);
    hipFree(v);
    hipFree(C);

    return 0;
}